/* Reference : The get neighbour is implemented by ourseleves, whereas the CUDA_BFS_KERNEL was referenced from https://github.com/siddharths2710/cuda_bfs 
				and the permutation generation code was referenced from https://www.geeksforgeeks.org/write-a-c-program-to-print-all-permutations-of-a-given-string/
*/ 
#include "hip/hip_runtime.h"


// #include <device_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

using namespace std;

#define NUM_NODES 24
#define N 2
#define COMB "1230"
#define STR "abcd"

typedef struct
{
	int start;     // Index of first adjacent node in Ea	
	int length;    // Number of adjacent nodes 
} Node;

typedef struct
{
	int id;
	string s;

} HashMap;

int count = 0;
int edgeNumber = 0;
HashMap map[10000000];
Node node[NUM_NODES];

// index value is  = NUM_NODES*(NUM_NODES-1);
int edges[NUM_NODES*(NUM_NODES-1)];
// #define NUM_PERMUTATIONS 



__global__ void CUDA_BFS_KERNEL(Node *Va, int *Ea, bool *Fa, bool *Xa, int *Ca,bool *done)
{

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id > NUM_NODES)
		return;


	if (Fa[id] == true && Xa[id] == false)
	{
		printf("%d ", id); //This printf gives the order of vertices in BFS	
		Fa[id] = false;
		Xa[id] = true;
		__syncthreads(); 
		int k = 0;
		int i;
		int start = Va[id].start;
		int end = start + Va[id].length;
		
		for (int i = start; i < end; i++) 
		{
			int nid = Ea[i];

			if (Xa[nid] == false)
			{
				Ca[nid] = Ca[id] + 1;
				Fa[nid] = true;
				*done = false;
			}
		}

	}

}

void swap(string a, int l, int i) 
{ 
	char temp; 
	temp = a[l];
	a[l] = a[i]; 
	a[i] = temp; 
} 


void permute(string a, int l, int r) 
{ 
   int i; 
   if (l == r) 
   {
	 // cout<<a<<endl;
		map[count].id = count;
		map[count].s = a;
		count+=1;
   }
   else
   { 
	   for (i = l; i <= r; i++) 
	   { 
		  	char temp; 
			temp = a[l];
			a[l] = a[i]; 
			a[i] = temp;
		  	permute(a, l+1, r); 
 
			temp = a[l];
			a[l] = a[i]; 
			a[i] = temp; //backtrack 
	   } 
   } 
}

void getneighbour(string s, int i)
{
	int mat[N][N];
	for(int j=0;j<N;j++)
	{
		for(int k=0;k<N;k++)
		{
			mat[j][k] = s[j*N+k]-'0';
		}
	}

	int posx,posy;
	for(int j=0;j<N;j++)
	{
		for(int k=0;k<N;k++)
		{
			if(mat[j][k] == 0)
			{
				posx = j;
				posy = k;
				break;
			}
		}
	}

	 if (posx == 0 && posy == 0) 
	 {
		int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		node[i].start = edgeNumber;
		node[i].length = 2;

		// Moving 0 to the right
		Temp[posx][posy] = Temp[posx][posy+1];
		Temp[posx][posy+1] = 0;

		string s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}


		// int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		// Moving 0 to the bottom
		Temp[posx][posy] = Temp[posx+1][posy];
		Temp[posx+1][posy] = 0;

		// s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}
	}


	if (posx == 0 && posy == 1) 
	{
		int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		node[i].start = edgeNumber;
		node[i].length = 2;

		// Moving 0 to the left
		Temp[posx][posy] = Temp[posx][posy-1];
		Temp[posx][posy-1] = 0;

		string s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}


		// int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		// Moving 0 to the bottom
		Temp[posx][posy] = Temp[posx+1][posy];
		Temp[posx+1][posy] = 0;

		// s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}
	}

	if (posx == 1 && posy == 0) 
	 {
		int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		node[i].start = edgeNumber;
		node[i].length = 2;

		// Moving 0 to the right
		Temp[posx][posy] = Temp[posx][posy+1];
		Temp[posx][posy+1] = 0;

		string s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}


		// int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		// Moving 0 to the top
		Temp[posx][posy] = Temp[posx-1][posy];
		Temp[posx-1][posy] = 0;

		// s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}
	}
	if (posx == 1 && posy == 1) 
	 {
		int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		node[i].start = edgeNumber;
		node[i].length = 2;

		// Moving 0 to the left
		Temp[posx][posy] = Temp[posx][posy-1];
		Temp[posx][posy-1] = 0;

		string s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}


		// int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		// Moving 0 to the top
		Temp[posx][posy] = Temp[posx-1][posy];
		Temp[posx-1][posy] = 0;

		// s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}
	}
	if (posy == 0) {

		int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		node[i].start = edgeNumber;
		node[i].length = 3;

		// Moving 0 to the top
		Temp[posx][posy] = Temp[posx-1][posy];
		Temp[posx-1][posy] = 0;

		string s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}


		// int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		// Moving 0 to the bottom
		Temp[posx][posy] = Temp[posx+1][posy];
		Temp[posx+1][posy] = 0;

		// s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}

		// int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		// Moving 0 to the right
		Temp[posx][posy] = Temp[posx][posy+1];
		Temp[posx][posy+1] = 0;

		// s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}
	}

	if (posy == N-1) {

		int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		node[i].start = edgeNumber;
		node[i].length = 3;

		// Moving 0 to the top
		Temp[posx][posy] = Temp[posx-1][posy];
		Temp[posx-1][posy] = 0;

		string s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}


		// int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		// Moving 0 to the bottom
		Temp[posx][posy] = Temp[posx+1][posy];
		Temp[posx+1][posy] = 0;

		// s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}

		// int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		// Moving 0 to the left
		Temp[posx][posy] = Temp[posx][posy-1];
		Temp[posx][posy-1] = 0;

		// s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}
	}

	if (posx == 0) {

		int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		node[i].start = edgeNumber;
		node[i].length = 3;

		// Moving 0 to the left
		Temp[posx][posy] = Temp[posx][posy-1];
		Temp[posx][posy-1] = 0;

		string s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}


		// int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		// Moving 0 to the right
		Temp[posx][posy] = Temp[posx][posy+1];
		Temp[posx][posy+1] = 0;

		// s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}

		// int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		// Moving 0 to the bottom
		Temp[posx][posy] = Temp[posx+1][posy];
		Temp[posx+1][posy] = 0;

		// s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}
	}

	if(posx == N-1) {
		int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		node[i].start = edgeNumber;
		node[i].length = 3;

		// Moving 0 to the left
		Temp[posx][posy] = Temp[posx][posy-1];
		Temp[posx][posy-1] = 0;

		string s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}


		// int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		// Moving 0 to the right
		Temp[posx][posy] = Temp[posx][posy+1];
		Temp[posx][posy+1] = 0;

		// s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}

		// int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		// Moving 0 to the top
		Temp[posx][posy] = Temp[posx-1][posy];
		Temp[posx-1][posy] = 0;

		// s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}
	}

	else {

		int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		node[i].start = edgeNumber;
		node[i].length = 4;

		// Moving 0 to the left
		Temp[posx][posy] = Temp[posx][posy-1];
		Temp[posx][posy-1] = 0;

		string s1 = STR;

		for(int j=0;j<2;j++)
		{
			for(int k=0;k<2;k++)
			{
				s1[j*2+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}


		// int Temp[2][2];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		// Moving 0 to the right
		Temp[posx][posy] = Temp[posx][posy+1];
		Temp[posx][posy+1] = 0;

		// s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}

		// int Temp[N][N];
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		// Moving 0 to the bottom
		Temp[posx][posy] = Temp[posx+1][posy];
		Temp[posx+1][posy] = 0;

		// s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				Temp[j][k] = mat[j][k];
			}
		}

		// Moving 0 to the top
		Temp[posx][posy] = Temp[posx-1][posy];
		Temp[posx-1][posy] = 0;

		// s1 = STR;

		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				s1[j*N+k] = Temp[j][k]+'0';
			}
		}

		for(int j=0;j<NUM_NODES;j++)
		{
			if (map[j].s == s1)
			{
				edges[edgeNumber++] = map[j].id;
			}
		}
	}
} 

// The BFS frontier corresponds to all the nodes being processed at the current level.

int main()
{

	permute(COMB, 0, N*N-1);

	for(int i=0;i<NUM_NODES;i++)
	{
		cout<<map[i].id<<" "<<map[i].s<<endl;
		// getneighbour(map[i].s, map[i].id);
	}

	for(int i=0;i<NUM_NODES;i++)
	{
		string s = map[i].s;
		int id = map[i].id;
		getneighbour(s, i);
	}

	cout<<"Done"<<endl;

	bool frontier[NUM_NODES] = { false };
	bool visited[NUM_NODES] = { false };
	int cost[NUM_NODES] = { 0 };

	int source = 0;
	frontier[source] = true;

	hipEvent_t start_kernel, stop_kernel;
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
	hipEventRecord(start_kernel);

	Node* Va;
	hipMalloc((void**)&Va, sizeof(Node)*NUM_NODES);
	hipMemcpy(Va, node, sizeof(Node)*NUM_NODES, hipMemcpyHostToDevice);

	int* Ea;
	hipMalloc((void**)&Ea, sizeof(int)*(NUM_NODES*(NUM_NODES+1)));
	hipMemcpy(Ea, edges, sizeof(int)*(NUM_NODES*(NUM_NODES+1)), hipMemcpyHostToDevice);

	bool* Fa;
	hipMalloc((void**)&Fa, sizeof(bool)*NUM_NODES);
	hipMemcpy(Fa, frontier, sizeof(bool)*NUM_NODES, hipMemcpyHostToDevice);

	bool* Xa;
	hipMalloc((void**)&Xa, sizeof(bool)*NUM_NODES);
	hipMemcpy(Xa, visited, sizeof(bool)*NUM_NODES, hipMemcpyHostToDevice);

	int* Ca;
	hipMalloc((void**)&Ca, sizeof(int)*NUM_NODES);
	hipMemcpy(Ca, cost, sizeof(int)*NUM_NODES, hipMemcpyHostToDevice);

	

	int num_blks = 1;
	int threads = 32;



	bool done;
	bool* d_done;
	hipMalloc((void**)&d_done, sizeof(bool));
	printf("\n\n");
	int count = 0;
    
    float runningTime = 0;

	printf("Order: \n\n");
	do 
	{
		count++;
		done = true;
		hipMemcpy(d_done, &done, sizeof(bool), hipMemcpyHostToDevice);
		
		CUDA_BFS_KERNEL <<<num_blks, threads >>>(Va, Ea, Fa, Xa, Ca,d_done);
		hipMemcpy(&done, d_done , sizeof(bool), hipMemcpyDeviceToHost);

	} while (done!=true);

	
	hipMemcpy(cost, Ca, sizeof(int)*NUM_NODES, hipMemcpyDeviceToHost);

	hipEventRecord(stop_kernel);
	hipEventSynchronize(stop_kernel);
	float k_time = 0;
    hipEventElapsedTime(&k_time, start_kernel, stop_kernel);
    // runningTime+=k_time;
    cout << "\nGPU TIME : " <<k_time <<"ms"<<" "<<"Level : "<<count<<std::endl<<std::endl;
		
	
	printf("Number of times the kernel is called : %d \n", count);


	printf("\nCost: ");
	for (int i = 0; i<NUM_NODES; i++)
		printf( "%d    ", cost[i]);
	printf("\n");
	// _getch();
	hipFree(Va);
	hipFree(Ea);
	hipFree(Fa);
	hipFree(Xa);
	hipFree(Xa);
	hipFree(d_done);
	
}
